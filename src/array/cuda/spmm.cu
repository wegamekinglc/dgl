#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./spmm.cuh"
#include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {

using namespace cuda;

namespace aten {
namespace {

/*! \brief Fill the vector started from ptr of size length with val */
template <typename DType>
void _Fill(DType* ptr, size_t length, DType val) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  int nt = FindNumThreads(length);
  int nb = (length + nt - 1) / nt;  // on x-axis, no need to worry about upperbound.
  CUDA_KERNEL_CALL(cuda::_FillKernel, nb, nt, 0, thr_entry->stream, ptr, length, val);
}

}  // namespace

namespace cusparse {

#if CUDART_VERSION < 11000
template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}
#endif

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

/*! Cusparse implementation of SpMM on Csr format. */
template <typename DType>
void CusparseCsrmm2(
    const DLContext& ctx,
    const CSRMatrix& csr,
    const DType* B_data, const DType* A_data,
    DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(ctx, m * n * sizeof(DType)));
  // all one data array
  DType* valptr = nullptr;
  if (!A_data) {
    valptr = static_cast<DType*>(device->AllocWorkspace(ctx, nnz * sizeof(DType)));
    _Fill(valptr, nnz, static_cast<DType>(1.));
  }
#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB, matC;
  constexpr auto cuda_dtype = std::is_same<DType, float>::value ? HIP_R_32F: HIP_R_64F;
  CUSPARSE_CALL(hipsparseCreateCsr(&matA,
      m, k, nnz,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      const_cast<DType*>(valptr? valptr : A_data),
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, cuda_dtype));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matB,
      n, k, n,
      const_cast<DType*>(B_data), cuda_dtype, HIPSPARSE_ORDER_COL));
  CUSPARSE_CALL(hipsparseCreateDnMat(&matC,
      m, n, m,
      trans_out, cuda_dtype, HIPSPARSE_ORDER_COL));

  auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  auto transB = HIPSPARSE_OPERATION_TRANSPOSE;
  size_t workspace_size;
  CUSPARSE_CALL(hipsparseSpMM_bufferSize(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      cuda_dtype, HIPSPARSE_CSRMM_ALG1,
      &workspace_size));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUSPARSE_CALL(hipsparseSpMM(
      thr_entry->cusparse_handle, transA, transB,
      &alpha, matA, matB, &beta, matC,
      cuda_dtype, HIPSPARSE_CSRMM_ALG1,
      workspace));
  device->FreeWorkspace(ctx, workspace);

  CUSPARSE_CALL(hipsparseDestroySpMat(matA));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matB));
  CUSPARSE_CALL(hipsparseDestroyDnMat(matC));
#else
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, (valptr)? valptr : A_data,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  CUSPARSE_CALL(hipsparseDestroyMatDescr(descr));
#endif
  if (valptr)
    device->FreeWorkspace(ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle)
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, thr_entry->stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(ctx, trans_out);
}
}  // namespace cusparse

#define SWITCH_OP(op, Op, ...)                                      \
  do {                                                              \
    if ((op) == "add") {                                            \
      typedef cuda::binary::Add<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "sub") {                                     \
      typedef cuda::binary::Sub<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "mul") {                                     \
      typedef cuda::binary::Mul<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "div") {                                     \
      typedef cuda::binary::Div<DType> Op;                          \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_lhs") {                                \
      typedef cuda::binary::CopyLhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else if ((op) == "copy_rhs") {                                \
      typedef cuda::binary::CopyRhs<DType> Op;                      \
      { __VA_ARGS__ }                                               \
    } else {                                                        \
      LOG(FATAL) << "Unsupported SpMM binary operator: " << op;     \
    }                                                               \
  } while (0)

/*!
 * \brief CUDA implementation of g-SpMM on Csr format.
 * \note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, typename DType>
void SpMMCsr(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const CSRMatrix& csr,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  int64_t feat_len = bcast.out_len;
  bool is_scalar_efeat = efeat.NumElements() == csr.indices->shape[0];
  bool use_efeat = op != "copy_lhs";

  if (reduce == "sum") {
    if ((!use_efeat || is_scalar_efeat) && feat_len > 64) {  // ge-spmm
      if (use_efeat && !IsNullArray(csr.data))  // reorder edge data
        efeat = IndexSelect(efeat, csr.data);
      SWITCH_OP(op, Op, {
        cuda::GESpMMCsr<IdType, DType, Op>(
          csr, ufeat, efeat, out, feat_len);
      });
    } else if (sizeof(IdType) == 4 && op == "copy_lhs") {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      cusparse::CusparseCsrmm2<DType>(
          ufeat->ctx, csr,
          static_cast<DType*>(ufeat->data),
          nullptr,
          static_cast<DType*>(out->data),
          x_length);
    } else if (sizeof(IdType) == 4 && op == "mul" && is_scalar_efeat) {  // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      if (!IsNullArray(csr.data))
        efeat = IndexSelect(efeat, csr.data);
      cusparse::CusparseCsrmm2<DType>(
          ufeat->ctx, csr,
          static_cast<DType*>(ufeat->data),
          static_cast<DType*>(efeat->data),
          static_cast<DType*>(out->data),
          x_length);
    } else {  // general kernel
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, NullArray(), NullArray());
      });
    }
  } else if (reduce == "max") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Max<IdType, DType> >(
          bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else if (reduce == "min") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Min<IdType, DType> >(
          bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

/*!
 * \brief CUDA implementation of g-SpMM on Coo format.
 */
template <int XPU, typename IdType, typename DType>
void SpMMCoo(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const COOMatrix& coo,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  if (reduce == "sum") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Sum<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, NullArray(), NullArray());
    });
  } else if (reduce == "max") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Max<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  }  else if (reduce == "min") {
    SWITCH_OP(op, Op, {
      cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Min<IdType, DType, true> > (
          bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template void SpMMCsr<kDLGPU, int32_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

template void SpMMCoo<kDLGPU, int32_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, float>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, double>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);

}  // namespace aten
}  // namespace dgl
